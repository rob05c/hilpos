#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include "hilpos.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>    
#include <vector>
#include <utility>

#include <linux/hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

using std::vector;
using std::pair;

using namespace hipcub; // debug

/// \todo fix to not be global
CachingDeviceAllocator g_allocator(true); // CUB caching allocator for device memory

/// \returns the device totalGlobalMem
inline size_t GetDeviceMemory() {
  hipDeviceProp_t properties;
  int deviceNum;
  CubDebugExit(hipGetDevice(&deviceNum));
  CubDebugExit(hipGetDeviceProperties(&properties, deviceNum));
  return properties.totalGlobalMem;
}

#ifndef MIN
#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#endif

#define GET_MID(MIN, MAX) (MIN + (MAX - MIN) / 2)

#define NEW_DIMENSION_MIN(VAL, MID, MIN, MAX) (MID * (VAL >= MID) + MIN * (VAL < MID))
#define NEW_DIMENSION_MAX(VAL, MID, MIN, MAX) (MID * (VAL < MID) + MAX * (VAL >= MID))

__global__ void k_create_hilbert_codes(cuda_star* stars, uint64_t* codes, size_t len,
                                          double xmin, double ymin, double zmin,
                                          double xmax, double ymax, double zmax) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= len)
    return; // skip the final block remainder

  const cuda_star& star = stars[i];
  const double x = star.x;
  const double y = star.y;
  const double z = star.z;

  uint64_t code = 0;
  #pragma unroll
  for(int i = 0, end = 21; i != end; ++i) {
    code = code << 3;

    const double xmid = GET_MID(xmin, xmax);
    const double ymid = GET_MID(ymin, ymax);
    const double zmid = GET_MID(zmin, zmax);

    const char bit0 = (y <= ymid);
    const char bit1 = (y <= ymid && z >= zmid) 
      || (y >= ymid && z <= zmid);
    const char bit2 = (x >= xmid && y <= ymid && z <= zmid)
      || (x <= xmid && y <= ymid && z >= zmid)
      || (x >= xmid && y >= ymid && z >= zmid)
      || (x <= xmid && y >= ymid && z <= zmid);
    const char bits = (bit0 << 2) | (bit1 << 1) | (bit2 << 0);
    code = code | bits;

    xmin = NEW_DIMENSION_MIN(x, xmid, xmin, xmax);
    xmax = NEW_DIMENSION_MAX(x, xmid, xmin, xmax);
    ymin = NEW_DIMENSION_MIN(y, ymid, ymin, ymax);
    ymax = NEW_DIMENSION_MAX(y, ymid, ymin, ymax);
    zmin = NEW_DIMENSION_MIN(z, zmid, zmin, zmax);
    zmax = NEW_DIMENSION_MAX(z, zmid, zmin, zmax);
  }
  codes[i] = code;
}

/// \return array of calculated hilbert codes of length len. Caller takes ownership.
uint64_t* create_hilbert_codes(cuda_star* stars, size_t len,
                               double xmin, double ymin, double zmin,
                               double xmax, double ymax, double zmax) {
  const unsigned int THREADS_PER_BLOCK = 512;
  cuda_star* cuda_stars;
  uint64_t*  cuda_codes;

  hipMalloc((void**)&cuda_stars, len * sizeof(cuda_star));
  hipMalloc((void**)&cuda_codes, len * sizeof(uint64_t));
  hipMemcpy(cuda_stars, stars, len * sizeof(cuda_star), hipMemcpyHostToDevice);
  k_create_hilbert_codes<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_stars, cuda_codes, len,
                                                                                                        xmin, ymin, zmin,
                                                                                                        xmax, ymax, zmax);
  uint64_t* codes = new uint64_t[len];
  hipMemcpy(codes, cuda_codes, len * sizeof(uint64_t), hipMemcpyDeviceToHost);
  hipFree(cuda_stars);
  hipFree(cuda_codes);
  return codes;
}

/// note: sorts the stars, according to the codes. Does NOT sort the codes, for efficiency. Easily could.
void cuda_sort(cuda_star* stars, uint64_t* codes, size_t len) {
  DoubleBuffer<uint64_t> d_keys;
  DoubleBuffer<cuda_star> d_values;
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(uint64_t) * len));
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(uint64_t) * len));
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(cuda_star) * len));
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(cuda_star) * len));

  CubDebugExit( hipMemcpy(d_keys.d_buffers[0], codes, sizeof(uint64_t) * len, hipMemcpyHostToDevice));
  CubDebugExit( hipMemcpy(d_values.d_buffers[0], stars, sizeof(cuda_star) * len, hipMemcpyHostToDevice));


  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, len));
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, len));

//  CubDebugExit( hipMemcpy(codes, d_keys.Current(), len * sizeof(uint64_t), hipMemcpyDeviceToHost));
  CubDebugExit( hipMemcpy(stars, d_values.Current(), len * sizeof(cuda_star), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));
}
